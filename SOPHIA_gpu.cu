//---------------------------------------------------------------------------------------------------
// SOPHIA_gpu Version 2.0: Smoothed Particle Hydrodynamics code In Advanced nuclear safety
// Developed by Eung Soo Kim, Young Beom Jo, So Hyun Park, Hae Yoon Choi in 2017
// ENERGy SYSTEM LABORATORY, NUCLEAR ENGINEERING DEPARTIMENT, SEOUL NATIONAL UNIVERSITY, SOUTH KOREA
//---------------------------------------------------------------------------------------------------
// Optimized by Dong Hak Lee, Yong Woo Sim in 2018 (2018.01.08)
// Copyright 2018(C) CoColink Inc.
//---------------------------------------------------------------------------------------------------
// Multi-GPU Optimized by Dong Hak Lee, Yong Woo Sim in 2019 (2019.01.09)
// Copyright 2019(C) CoColink Inc.
//---------------------------------------------------------------------------------------------------
// Code Restructured by Eung Soo Kim, Hee Sang Yoo, Young Beom Jo, Hae Yoon Choi, Su-San Park, Jin Woo Kim, Yelyn Ahn, Tae Soo Choi in (2019.08.08)
// Copyright 2019(C) ESLAB, SEOUL NATIONAL UNIVERSITY, SOUTH KOREA.
//---------------------------------------------------------------------------------------------------

#include <stdio.h>
#include <string>
#include <algorithm>
#include <math.h>
#include <time.h>
#include <pthread.h>
#include <hipcub/hipcub.hpp>
#include <float.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include ""
// #include "hip/device_functions.h"
#include "Cuda_Error.cuh"

#include "Variable_Type.cuh"
#include "Parameters.cuh"
#include "class_Cuda_Particle_Array.cuh"


//---------------------------------------------------------------
// 전역변수 선언
//---------------------------------------------------------------

// Solver 파라미터 선언 (function_init.cuh)_______
// host
int_t vii[vii_size];
Real vif[vif_size];

// device - solver options
__constant__ int_t k_vii[vii_size];
__constant__ Real k_vif[vif_size];

// device - table
__constant__ Real k_Tab_T[table_size]; 	//table_size는 Parameter.cuh에 정의됨
__constant__ Real k_Tab_h[table_size];
__constant__ Real k_Tab_k[table_size];
__constant__ Real k_Tab_cp[table_size];
__constant__ Real k_Tab_vis[table_size];
__constant__ int k_table_index[10]; 		// table의 시작 주소
__constant__ int k_table_size[10]; 	 		// table별 데이터 수

// 입자 선언 _________
// for main domain
part1*HP1;							// host 전체입자
part1*DHP1[Max_GPU];		// gpu당 할당할 host 입자

// for data exchange
part1*send_P1[Max_GPU],*send_rSP1[Max_GPU],*send_lSP1[Max_GPU],*recv_P1[Max_GPU];
p2p_part3*send_P3[Max_GPU],*send_rSP3[Max_GPU],*send_lSP3[Max_GPU],*recv_P3[Max_GPU];

// table - host
Real host_Tab_T[table_size];
Real host_Tab_h[table_size];
Real host_Tab_k[table_size];
Real host_Tab_cp[table_size];
Real host_Tab_vis[table_size];

int host_table_index[10];
int host_table_size[10];

// 병렬 분기 _________
hipStream_t str1[Max_GPU];
hipStream_t str2[Max_GPU];
pthread_barrier_t barrier;

// Open Boundary _______
Real space;				  // open boundary inlet을 위한 가상 격자의 간격
int Nsx=0;					// x축 방향 격자 수
int Nsy=0;
int Nsz=0;					// y축 방향 격자 수
int buffer_size=0;  // 생성 입자룰 위한 여분의 메모리 크기

__device__ int num_buffer[1];	// buffer 입자의 총 수를 저장하는 전역 변수 (Open Boundary/APR 모델 사용시 필요)

// Plot Data ______
int num_plot_data;				// plot data 개수
char plot_data[20][20]; 	// plot 할 변수


//---------------------------------------------------------------


#include "function_init.cuh"
#include "functions_NNPS.cuh"
#include "functions_KNL.cuh"
#include "functions_PROP.cuh"
#include "functions_PREP.cuh"
#include "functions_IBM.cuh"
#include "functions_INTERACTION.cuh"
#include "functions_ENERGY.cuh"
#include "functions_TIME.cuh"
#include "functions_PPE.cuh"
#include "functions_OUTPUT.cuh"
#include "functions_BC.cuh"
#include "functions_MOST.cuh"
// #include "functions_APR.cuh"
//#include "functions_APR_etc.cuh"
#include "functions_TIME_ISPH.cuh"
#include "ISPH_Calc.cuh"
#include "ISPH.cuh"

////////////////////////////////////////////////////////////////////////
int main(int argc,char**argv)
{
	memset(vii,0,sizeof(int_t)*vii_size);
	memset(vif,0,sizeof(Real)*vif_size);

	ngpu=atoi(argv[1]);

	char fn[64],fn2[64];
	strcpy(fn,"./input/solv.txt");
	strcpy(fn2,"./input/data.txt");

	read_solv_input(vii,vif,fn);
	read_table(fn2);

	switch(solver_type){
		case Isph:
			ISPH(vii,vif);
			break;
		default:
			ISPH(vii,vif);
	}

	return 0;
}
